
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
    hipDeviceProp_t prop;
    int device_id;

    hipGetDevice( &device_id);

    printf("Current Device ID: %d\n", device_id);

    memset( &prop, 0, sizeof( hipDeviceProp_t)); // copy 0 to cudaDeviceProp object or initialization.
    prop.major = 9;
    prop.minor = 1;

    hipChooseDevice( &device_id, &prop); // device_id and prop will be reassigned by this function with pass by reference. 
    printf("CUDA DEVICE ID that best matches the requirement: %d\n", device_id);

    hipSetDevice (device_id); // set device_id as where host threads will be executed.

}